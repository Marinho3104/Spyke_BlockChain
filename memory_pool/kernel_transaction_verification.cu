#include "hip/hip_runtime.h"


#include "kernel_transaction_verification.cuh"
#include "kernel_block_part_verification.cuh"
#include "memory_pool_initialization.cuh"
#include "transaction_definitions.h"
#include "utils_functions.h"
#include "transaction.h"
#include "block_part.h"

#include <iostream>

// Nvcc libs
#include <cuda/semaphore>
#include <hip/hip_runtime.h>
#include <stdio.h>

namespace memory_pool::cuda {

    hipStream_t transaction_verification_stream;

    void* transaction_verification_data;
    void* transaction_verification_previous_data;

    ::cuda::std::binary_semaphore* transaction_verification_previous_data_semphores;

    ::cuda::std::binary_semaphore* transaction_verification_semaphores;
    ::cuda::std::binary_semaphore* transaction_verification_thread_ready;

}

void memory_pool::cuda::set_previous_data() {

    char* _file_path = 
        ( char* ) malloc( sizeof( MEMORY_POOL_KERNEL_BLOCK_PART_VERIFICATION_STORE_DIRECTORY ) + 128 );

    void* _hash_hex;

    memcpy(
        _file_path,
        MEMORY_POOL_KERNEL_BLOCK_PART_VERIFICATION_STORE_DIRECTORY,
        sizeof( MEMORY_POOL_KERNEL_BLOCK_PART_VERIFICATION_STORE_DIRECTORY ) - 1
    );

    _file_path[ sizeof( MEMORY_POOL_KERNEL_BLOCK_PART_VERIFICATION_STORE_DIRECTORY ) + 127 ] = 0; 

    void* _current_block_hash_search;

    while( 1 ) {

        _current_block_hash_search = current_block_part_hash;

        for ( int _ = 0; _ < MEMORY_POOL_KERNEL_TRANSACTION_VERIFICATION_BLOCKS_TIMES_BLOCK_THREADS; _++ ) {
        
            if ( ! transaction_verification_thread_ready[ _ ].try_acquire() ) {

                while( 1 ) {

                    std::cout << "Search loop" << std::endl;

                    _hash_hex = 
                        utils::convert_bytes_hex( _current_block_hash_search, 64 );

                    std::cout << ( char* ) _hash_hex << std::endl;

                    memcpy(
                        _file_path + sizeof( MEMORY_POOL_KERNEL_BLOCK_PART_VERIFICATION_STORE_DIRECTORY ) - 1,
                        _hash_hex,
                        128
                    ); free( _hash_hex );   

                    size_t _file_data_size;

                    void* _block_part_data = 
                        utils::get_file_data( _file_path, _file_data_size );

                    if ( ! _block_part_data ) { std::cout << "Block from file null " << std::endl; exit( 1 ); }

                    types::Block_Part* _block_part = 
                        ( types::Block_Part* ) _block_part_data;

                    _current_block_hash_search = _block_part->previous_hash;

                    void* _previous_data = 
                        _block_part->get_transaction_data(
                            transaction_verification_data + 64
                        );

                    if ( ! _previous_data ) { std::cout << "Transaction data null " << std::endl; continue; }

                    memcpy(
                        transaction_verification_previous_data + _ * TRANSACTION_LENGTH,
                        _previous_data,
                        TRANSACTION_LENGTH
                    );

                    transaction_verification_previous_data_semphores->release();

                    break;

                }

            }

            else transaction_verification_thread_ready[ _ ].release();
        
        }

        sleep( 2 );

    }

}

__global__ void memory_pool::cuda::kernel_transaction_verification( void* __data, ::cuda::std::binary_semaphore* __semaphore, ::cuda::std::binary_semaphore* __semaphore_ready, void* __previous_data, ::cuda::std::binary_semaphore* __previous_data_semaphores, void* __memory_pool, ::cuda::std::binary_semaphore* __memory_pool_sems, ::cuda::std::binary_semaphore* __memory_pool_broudcast_sems,  uint64_t* __ready_transactions, uint64_t* __memory_pool_max_transactions_capacity, unsigned char* __public_key_type, bool* __public_key_type_enable, unsigned char* __block_division, uint32_t* __size_span_public_key ) {

    int _global_id = 
        blockIdx.x * blockDim.x + threadIdx.x;

    while( 1 ) {

        // printf("To be Acquired %d\n", _global_id );

        __semaphore[ _global_id ].acquire();

        printf("Acquired %d\n", _global_id );

        printf("Waiting for previous data %d\n", _global_id );

        __previous_data_semaphores->acquire();

        printf("Acquired previous data %d\n", _global_id );

        // printf("Public key enable %d\n", *__public_key_type_enable );

        // Checks if transaction is already in pool
        if ( ! kernel_memory_pool_transaction_check( __data + _global_id * TRANSACTION_PROPAGATION_LENGTH, __memory_pool, __memory_pool_max_transactions_capacity ) ) {

            // Make all needed confirmations TODO

            types::Transaction* _previous_transaction = 
                ( types::Transaction* ) ( __previous_data + _global_id * TRANSACTION_LENGTH );

            types::Transaction* _transaction = 
                ( types::Transaction* ) ( __data + _global_id * TRANSACTION_PROPAGATION_LENGTH );

            if (
                _previous_transaction->from_balance >= _transaction->amount + _transaction->fee
            ) {

                // If all confirmations succeed

                // Checks if public key type is enable
                if ( ! *__public_key_type_enable || kernel_get_public_key_type( __data + _global_id * TRANSACTION_PROPAGATION_LENGTH + WALLET_WALLET_DEFINITIONS_ED25519_SIGNATURE_LENGTH, __block_division, __size_span_public_key ) == *__public_key_type ) 
                
                    kernel_store_transaction_data( __data + _global_id * TRANSACTION_PROPAGATION_LENGTH, __memory_pool, __memory_pool_sems, __memory_pool_broudcast_sems, __ready_transactions, __memory_pool_max_transactions_capacity );

            }

        } // else printf( "Already in memory pool" );

        __semaphore_ready[ _global_id ].release();

    }

}

__device__ void memory_pool::cuda::kernel_store_transaction_data( void* __transaction_data, void* __memory_pool, ::cuda::std::binary_semaphore* __memory_pool_sems,  ::cuda::std::binary_semaphore* __memory_pool_broudcast_sems, uint64_t* __ready_transactions, uint64_t* __memory_pool_max_transactions_capacity ) {

    for (
        uint64_t _ = 0; _ < *__memory_pool_max_transactions_capacity; _++
    ) {

        if ( __memory_pool_sems[ _ ].try_acquire() ) {

            printf("Store in memory pool %d\n", _);

            memcpy(
                __memory_pool + _ * TRANSACTION_PROPAGATION_LENGTH,
                __transaction_data,
                TRANSACTION_PROPAGATION_LENGTH
            );

            __memory_pool_broudcast_sems[ _ ].release();

            atomicAdd( ( unsigned long long* ) __ready_transactions, 1 );
  
            break;

        }

    }

}

__device__ bool memory_pool::cuda::kernel_memory_pool_transaction_check( void* __transaction_data, void* __memory_pool, uint64_t* __memory_pool_max_transactions_capacity ) {

    bool _rtr;

    for (
        uint64_t _ = 0; _ < *__memory_pool_max_transactions_capacity; _++
    ) {

        _rtr = 1;

        for ( int __ = 0; __ < TRANSACTION_PROPAGATION_LENGTH; __++ )

            if ( *( unsigned char* ) ( __memory_pool + _ * TRANSACTION_PROPAGATION_LENGTH + __ ) != *( unsigned char* ) ( __transaction_data + __ ) ) { _rtr = 0; break; }

        if ( _rtr ) return _rtr;

    }

    return 0;

}

__device__ unsigned char memory_pool::cuda::kernel_get_public_key_type( void* __public_key, unsigned char* __block_division, uint32_t* __size_span_public_key ) {

    uint32_t* _pk_value =   
        ( uint32_t* ) __public_key;

    unsigned char _type = 0;

    for (; _type < *__block_division - 1; _type++ )

        if ( *_pk_value < ( _type + 1 ) * *__size_span_public_key ) break;

    return _type;

}




