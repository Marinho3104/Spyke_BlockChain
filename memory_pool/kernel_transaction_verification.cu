#include "hip/hip_runtime.h"


#include "kernel_transaction_verification.cuh"
#include "memory_pool_initialization.cuh"
#include "transaction_definitions.h"
// #include "fs_calls.cu.h"

// Nvcc libs
#include <cuda/semaphore>
#include <hip/hip_runtime.h>
#include <stdio.h>

namespace memory_pool::cuda {

    hipStream_t transaction_verification_stream;

    void* transaction_verification_data;

    ::cuda::std::binary_semaphore* transaction_verification_semaphores;
    ::cuda::std::binary_semaphore* transaction_verification_thread_ready;

}

__global__ void memory_pool::cuda::kernel_transaction_verification( void* __data, ::cuda::std::binary_semaphore* __semaphore, ::cuda::std::binary_semaphore* __semaphore_ready, void* __memory_pool, ::cuda::std::binary_semaphore* __memory_pool_sems, ::cuda::std::binary_semaphore* __memory_pool_broudcast_sems,  uint64_t* __ready_transactions, uint64_t* __memory_pool_max_transactions_capacity, unsigned char* __public_key_type, bool* __public_key_type_enable ) {

    int _global_id = 
        blockIdx.x * blockDim.x + threadIdx.x;

    while( 1 ) {

        // printf("To be Acquired %d\n", _global_id );

        __semaphore[ _global_id ].acquire();

        printf("Acquired %d\n", _global_id );

        // Checks if transaction is already in pool
        if ( ! kernel_memory_pool_transaction_check( __data + _global_id * TRANSACTION_PROPAGATION_LENGTH, __memory_pool, __memory_pool_max_transactions_capacity ) ) {

            // Make all needed confirmations TODO


            // If all confirmations succed

            // Checks if public key type is enable
            if ( *__public_key_type_enable ) {

            }

            else kernel_store_transaction_data( __data + _global_id * TRANSACTION_PROPAGATION_LENGTH, __memory_pool, __memory_pool_sems, __memory_pool_broudcast_sems, __ready_transactions, __memory_pool_max_transactions_capacity );

        }

        __semaphore_ready[ _global_id ].release();

    }

}

__device__ void memory_pool::cuda::kernel_store_transaction_data( void* __transaction_data, void* __memory_pool, ::cuda::std::binary_semaphore* __memory_pool_sems,  ::cuda::std::binary_semaphore* __memory_pool_broudcast_sems, uint64_t* __ready_transactions, uint64_t* __memory_pool_max_transactions_capacity ) {

    for (
        uint64_t _ = 0; _ < *__memory_pool_max_transactions_capacity; _++
    ) {

        if ( __memory_pool_sems[ _ ].try_acquire() ) {

            memcpy(
                __memory_pool + _ * TRANSACTION_PROPAGATION_LENGTH,
                __transaction_data,
                TRANSACTION_PROPAGATION_LENGTH
            );

            __memory_pool_broudcast_sems[ _ ].release();

            atomicAdd( ( int* ) __ready_transactions, 1 );
  
            break;

        }

    }

}

__device__ bool memory_pool::cuda::kernel_memory_pool_transaction_check( void* __transaction_data, void* __memory_pool, uint64_t* __memory_pool_max_transactions_capacity ) {

    bool _rtr;

    for (
        uint64_t _ = 0; _ < *__memory_pool_max_transactions_capacity; _++
    ) {

        _rtr = 1;

        for ( int __ = 0; __ < TRANSACTION_PROPAGATION_LENGTH; __++ )

            if ( *( unsigned char* ) ( __memory_pool + _ * TRANSACTION_PROPAGATION_LENGTH + __ ) != *( unsigned char* ) ( __transaction_data + __ ) ) { _rtr = 0; break; }

        if ( _rtr ) return _rtr;

    }

    return 0;

}





