#include "hip/hip_runtime.h"


#include "kernel_transaction_verification.cuh"
#include "memory_pool_initialization.cuh"
#include "transaction_definitions.h"
// #include "fs_calls.cu.h"

// Nvcc libs
#include <cuda/semaphore>
#include <hip/hip_runtime.h>
#include <stdio.h>

namespace memory_pool::cuda {

    hipStream_t transaction_verification_stream;

    void* transaction_verification_data;

    ::cuda::std::binary_semaphore* transaction_verification_semaphores;
    ::cuda::std::binary_semaphore* transaction_verification_thread_ready;

}

__global__ void memory_pool::cuda::kernel_transaction_verification( void* __data, ::cuda::std::binary_semaphore* __semaphore, ::cuda::std::binary_semaphore* __semaphore_ready, void* __memory_pool, ::cuda::std::binary_semaphore* __memory_pool_sems, ::cuda::std::binary_semaphore* __memory_pool_broudcast_sems,  uint64_t* __ready_transactions, uint64_t* __memory_pool_max_transactions_capacity, unsigned char* __public_key_type, bool* __public_key_type_enable, unsigned char* __block_division, uint32_t* __size_span_public_key ) {

    int _global_id = 
        blockIdx.x * blockDim.x + threadIdx.x;

    while( 1 ) {

        // printf("To be Acquired %d\n", _global_id );

        __semaphore[ _global_id ].acquire();

        printf("Acquired %d\n", _global_id );
        printf("Public key enable %d\n", *__public_key_type_enable );

        // Checks if transaction is already in pool
        if ( ! kernel_memory_pool_transaction_check( __data + _global_id * TRANSACTION_PROPAGATION_LENGTH, __memory_pool, __memory_pool_max_transactions_capacity ) ) {

            // Make all needed confirmations TODO


            // If all confirmations succed

            // Checks if public key type is enable
            if ( ! *__public_key_type_enable || kernel_get_public_key_type( __data + _global_id * TRANSACTION_PROPAGATION_LENGTH + WALLET_WALLET_DEFINITIONS_ED25519_SIGNATURE_LENGTH, __block_division, __size_span_public_key ) == *__public_key_type ) 
            
                kernel_store_transaction_data( __data + _global_id * TRANSACTION_PROPAGATION_LENGTH, __memory_pool, __memory_pool_sems, __memory_pool_broudcast_sems, __ready_transactions, __memory_pool_max_transactions_capacity );

        }

        __semaphore_ready[ _global_id ].release();

    }

}

__device__ void memory_pool::cuda::kernel_store_transaction_data( void* __transaction_data, void* __memory_pool, ::cuda::std::binary_semaphore* __memory_pool_sems,  ::cuda::std::binary_semaphore* __memory_pool_broudcast_sems, uint64_t* __ready_transactions, uint64_t* __memory_pool_max_transactions_capacity ) {

    for (
        uint64_t _ = 0; _ < *__memory_pool_max_transactions_capacity; _++
    ) {

        if ( __memory_pool_sems[ _ ].try_acquire() ) {

            printf("Store in memory pool %d\n", _);

            memcpy(
                __memory_pool + _ * TRANSACTION_PROPAGATION_LENGTH,
                __transaction_data,
                TRANSACTION_PROPAGATION_LENGTH
            );

            __memory_pool_broudcast_sems[ _ ].release();

            atomicAdd( ( unsigned long long* ) __ready_transactions, 1 );
  
            break;

        }

    }

}

__device__ bool memory_pool::cuda::kernel_memory_pool_transaction_check( void* __transaction_data, void* __memory_pool, uint64_t* __memory_pool_max_transactions_capacity ) {

    bool _rtr;

    for (
        uint64_t _ = 0; _ < *__memory_pool_max_transactions_capacity; _++
    ) {

        _rtr = 1;

        for ( int __ = 0; __ < TRANSACTION_PROPAGATION_LENGTH; __++ )

            if ( *( unsigned char* ) ( __memory_pool + _ * TRANSACTION_PROPAGATION_LENGTH + __ ) != *( unsigned char* ) ( __transaction_data + __ ) ) { _rtr = 0; break; }

        if ( _rtr ) return _rtr;

    }

    return 0;

}

__device__ unsigned char memory_pool::cuda::kernel_get_public_key_type( void* __public_key, unsigned char* __block_division, uint32_t* __size_span_public_key ) {

    uint32_t* _pk_value =   
        ( uint32_t* ) __public_key;

    unsigned char _type = 0;

    for (; _type < *__block_division - 1; _type++ )

        if ( *_pk_value < ( _type + 1 ) * *__size_span_public_key ) break;

    return _type;

}




