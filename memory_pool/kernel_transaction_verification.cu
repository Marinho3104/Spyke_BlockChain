#include "hip/hip_runtime.h"


#include "kernel_transaction_verification.cuh"
#include "memory_pool_initialization.cuh"
#include "transaction_definitions.h"

// Nvcc libs
#include <cuda/semaphore>
#include <hip/hip_runtime.h>
#include <stdio.h>

namespace memory_pool::cuda {

    hipStream_t transaction_verification_stream;

    void* transaction_verification_data;

    ::cuda::std::binary_semaphore* transaction_verification_semaphores;
    ::cuda::std::binary_semaphore* transaction_verification_thread_ready;

}

__global__ void memory_pool::cuda::kernel_transaction_verification( void* __data, ::cuda::std::binary_semaphore* __semaphore, ::cuda::std::binary_semaphore* __semaphore_ready, void* __memory_pool, ::cuda::std::binary_semaphore* __memory_pool_sems, uint64_t* __ready_transactions, uint64_t* __memory_pool_max_transactions_capacity ) {

    int _global_id = 
        blockIdx.x * blockDim.x + threadIdx.x;

    while( 1 ) {

        printf("To be Acquired %d\n", _global_id );

        __semaphore[ _global_id ].acquire();

        printf("Acquired %d\n", _global_id );

        // Make all needed confirmations TODO

        // If all confirmations succed
        
        kernel_store_transaction_data( __data + _global_id * TRANSACTION_PROPAGATION_LENGTH, __memory_pool, __memory_pool_sems, __ready_transactions, __memory_pool_max_transactions_capacity );

        __semaphore_ready[ _global_id ].release();

    }

}

__device__ void memory_pool::cuda::kernel_store_transaction_data( void* __transaction_data, void* __memory_pool, ::cuda::std::binary_semaphore* __memory_pool_sems, uint64_t* __ready_transactions, uint64_t* __memory_pool_max_transactions_capacity ) {

    for (
        uint64_t _ = 0; _ < *__memory_pool_max_transactions_capacity; _++
    ) {

        if ( __memory_pool_sems[ _ ].try_acquire() ) {

            memcpy(
                __memory_pool + _ * TRANSACTION_PROPAGATION_LENGTH,
                __transaction_data,
                TRANSACTION_PROPAGATION_LENGTH
            );
            
            break;

        }

    }

}






