
#include "kernel_transaction_verification.cuh" // Transaction verification functions
#include "kernel_block_part_verification.cuh" // Block part functions
#include "kernel_initializations.cuh" // Extern functions
#include "memory_pool_initialization.cuh"
#include "utils_functions_cuda.cuh"

// Nvcc libs
#include <cuda/semaphore>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>



extern "C++" {

    void memory_pool::cuda::launch_kernel_transaction_verification() {

        // Initialize stream
        hipStreamCreate( &transaction_verification_stream ); utils::cuda::check_cuda_error();

        hipMallocManaged( &transaction_verification_semaphores, MEMORY_POOL_KERNEL_TRANSACTION_VERIFICATION_ALL_THREADS_SEMAPHORES_SIZE ); utils::cuda::check_cuda_error();
        hipMallocManaged( &transaction_verification_thread_ready, MEMORY_POOL_KERNEL_TRANSACTION_VERIFICATION_ALL_THREADS_SEMAPHORES_SIZE ); utils::cuda::check_cuda_error();

        for ( int _ = 0; _ < MEMORY_POOL_KERNEL_TRANSACTION_VERIFICATION_BLOCKS_TIMES_BLOCK_THREADS; _++ ) {

            new ( transaction_verification_semaphores + _ ) ::cuda::std::binary_semaphore( 0 );
            new ( transaction_verification_thread_ready + _ ) ::cuda::std::binary_semaphore( 1 );

        }

        hipMallocManaged( &transaction_verification_data, MEMORY_POOL_KERNEL_TRANSACTION_VERIFICATION_ALL_THREADS_TRANSACTION_SIZE ); utils::cuda::check_cuda_error();

        // Launch function
        kernel_transaction_verification 
            <<< MEMORY_POOL_KERNEL_TRANSACTION_VERIFICATION_BLOCKS, MEMORY_POOL_KERNEL_TRANSACTION_VERIFICATION_BLOCK_THREADS, 0, transaction_verification_stream >>> 
                ( transaction_verification_data, transaction_verification_semaphores, transaction_verification_thread_ready, memory_pool, memory_pool_sems, memory_pool_data_broudcast_sems, ready_transactions_count, memory_pool_transaction_capacity, public_key_type, public_key_type_enable, block_division, size_span_block_division );

        utils::cuda::check_cuda_error();

    }

    void memory_pool::cuda::launch_kernel_block_part_verification() {

        // Initialize stream
        hipStreamCreate( &block_part_stream ); utils::cuda::check_cuda_error();

        hipMallocManaged( &block_part_verification_semaphores, MEMORY_POOL_KERNEL_BLOCK_PART_VERIFICATION_ALL_THREADS_SEMAPHORES_SIZE ); utils::cuda::check_cuda_error();
        hipMallocManaged( &block_part_verification_thread_ready, MEMORY_POOL_KERNEL_BLOCK_PART_VERIFICATION_ALL_THREADS_SEMAPHORES_SIZE ); utils::cuda::check_cuda_error();

        for ( int _ = 0; _ < MEMORY_POOL_KERNEL_BLOCK_PART_VERIFICATION_ALL_THREADS_SEMAPHORES_SIZE; _++ ) {

            new ( block_part_verification_semaphores + _ ) ::cuda::std::binary_semaphore( 0 );
            new ( block_part_verification_thread_ready + _ ) ::cuda::std::binary_semaphore( 1 );

        }

        hipMallocManaged( &block_part_verification_data, MEMORY_POOL_KERNEL_BLOCK_PART_VERIFICATION_DATA ); utils::cuda::check_cuda_error();

        for ( int _ = 0; _ < MEMORY_POOL_KERNEL_BLOCK_PART_VERIFICATION_DATA; _++ ) block_part_verification_data[ _ ] = 0;

        kernel_block_part_verification
            <<< MEMORY_POOL_KERNEL_BLOCK_PART_VERIFICATION_BLOCKS, MEMORY_POOL_KERNEL_BLOCK_PART_VERIFICATION_BLOCK_THREADS, 0, block_part_stream >>>
                ( block_part_verification_data, block_part_verification_semaphores );

        utils::cuda::check_cuda_error();

    }


}