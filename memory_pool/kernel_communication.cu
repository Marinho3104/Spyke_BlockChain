

#include "kernel_transaction_verification.cuh" // Transaction verification variables
#include "kernel_communication.cuh" // Kernel communication functions
#include "transaction_definitions.h" // Transaction definitions
#include "memory_pool_initialization.cuh"
#include "block_part.h"
#include "connection.h" // Struct Connection
#include "propagation_protocol_definitions.h"
#include "propagation_protocol.h" // Struct Propagation Protocol
#include "packet.h" // Struct Packet

// Nvcc libs
#include <cuda/semaphore>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

void memory_pool::cuda::handle_block_part( void* __block_part_data ) {

    printf("Block part received in CPU cuda\n");

    types::Block_Part* _block_part = 
        ( types::Block_Part* ) __block_part_data;

    void* _data = 
        _block_part->get_transaction_data() + WALLET_WALLET_DEFINITIONS_ED25519_SIGNATURE_LENGTH + WALLET_WALLET_DEFINITIONS_ED25519_PUBLIC_KEY_LENGTH + WALLET_WALLET_DEFINITIONS_ED25519_PUBLIC_KEY_LENGTH;

    std::cout << "Transaction count: " << _block_part->transactions_count  << std::endl; 

    if ( ! _block_part->transactions_count ) return;

    std::cout << "Transaction amount: " << *( uint64_t* ) _data  << std::endl; 
    std::cout << "Transaction fee: " << *( uint64_t* ) ( _data + 8 )  << std::endl; 
    std::cout << "Transaction nonce: " << *( uint64_t* ) ( _data + 8 + 8 ) << std::endl; 

}

void memory_pool::cuda::handle_transaction( void* __transaction_data ) {

    for (
        int _ = 0;
        _ < MEMORY_POOL_KERNEL_TRANSACTION_VERIFICATION_BLOCKS_TIMES_BLOCK_THREADS;
        _++
    ) {

        // Thread was available
        if ( transaction_verification_thread_ready[ _ ].try_acquire() ) {

            // Copies the transaction data into device memory
            memcpy(
                transaction_verification_data + _ * (TRANSACTION_PROPAGATION_LENGTH),
                __transaction_data,
                TRANSACTION_PROPAGATION_LENGTH
            );

            transaction_verification_semaphores[ _ ].release(); break;

        }

    }

}

void memory_pool::cuda::transactions_to_broudcast( p2p::Connection** __connections, uint16_t __connections_count ) {

    p2p::Propagation_Protocol* _propagation_protocol = 
        ( p2p::Propagation_Protocol* ) malloc( sizeof( p2p::Propagation_Protocol ) );

    p2p::Packet* _packet;

    for (
        int _ = 0;
        _ < *memory_pool_transaction_capacity;
        _++
    ) {
        
        // Thread was available
        if ( memory_pool_data_broudcast_sems[ _ ].try_acquire() ) {
            
            new ( _propagation_protocol ) 
                p2p::Propagation_Protocol(
                    P2P_PROTOCOLS_PROPAGATION_PROTOCOL_DEFINITIONS_PROPAGATION_TYPE_TRANSACTION,
                    TRANSACTION_PROPAGATION_LENGTH,
                    memory_pool + _ * (TRANSACTION_PROPAGATION_LENGTH)
                );

            _packet = 
                _propagation_protocol->get_packet();

            for ( uint16_t _ = 0; _ < __connections_count; _++ )

                if ( __connections[ _ ]  ) __connections[ _ ]->send_packet( _packet );

        }

    }

}

void* memory_pool::cuda::get_transaction_data_from_memory_pool( uint32_t* __transactions_count ) {

    uint64_t _ready_transactions = *ready_transactions_count;

    *__transactions_count = _ready_transactions;

    if ( ! *__transactions_count ) return 0;

    void* _data = 
        malloc( _ready_transactions * (TRANSACTION_LENGTH) ), *_rtr = _data;

    for ( uint64_t _ = 0; _ready_transactions && _ < *memory_pool_transaction_capacity; _++ )

        // Have data
        if ( ! memory_pool_sems[ _ ].try_acquire() ) {

            memset(
                _data,
                0,
                TRANSACTION_LENGTH
            );

            memcpy(
                _data,
                memory_pool + _ * (TRANSACTION_PROPAGATION_LENGTH),
                TRANSACTION_PROPAGATION_LENGTH
            );
            
            _data = _data + TRANSACTION_LENGTH;

            memory_pool_sems[ _ ].release();

            (*ready_transactions_count)--; _ready_transactions--;

        }

        // No data
        else memory_pool_sems[ _ ].release();

    return _rtr;

}

