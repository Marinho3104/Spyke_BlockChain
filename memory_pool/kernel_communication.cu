

#include "kernel_transaction_verification.cuh" // Transaction verification variables
#include "kernel_communication.cuh" // Kernel communication functions
#include "transaction_definitions.h" // Transaction definitions

// Nvcc libs
#include <cuda/semaphore>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

void memory_pool::cuda::handle_transaction( void* __transaction_data ) {

    for (
        int _ = 0;
        _ < MEMORY_POOL_KERNEL_TRANSACTION_VERIFICATION_BLOCKS_TIMES_BLOCK_THREADS;
        _++
    ) {

        // Thread was available
        if ( transaction_verification_thread_ready[ _ ].try_acquire() ) {

            // Copies the transaction data into device memory
            memcpy(
                transaction_verification_data + _ * TRANSACTION_PROPAGATION_LENGTH,
                __transaction_data,
                TRANSACTION_PROPAGATION_LENGTH
            );

            transaction_verification_semaphores[ _ ].release(); break;

        }

    }

}
