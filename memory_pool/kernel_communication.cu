

#include "kernel_transaction_verification.cuh" // Transaction verification variables
#include "utils_functions_cuda.cuh"
#include "kernel_block_part_verification.cuh"
#include "kernel_communication.cuh" // Kernel communication functions
#include "transaction_definitions.h" // Transaction definitions
#include "memory_pool_initialization.cuh"
#include "utils_functions.h"
#include "block_part.h"
#include "connection.h" // Struct Connection
#include "propagation_protocol_definitions.h"
#include "propagation_protocol.h" // Struct Propagation Protocol
#include "packet.h" // Struct Packet

// Nvcc libs
#include <cuda/semaphore>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

void memory_pool::cuda::handle_block_part( void* __block_part_data, uint64_t __block_part_data_size ) {

    char* _file_path = 
        ( char* ) malloc( sizeof( MEMORY_POOL_KERNEL_BLOCK_PART_VERIFICATION_STORE_DIRECTORY ) + 129 );

    _file_path[ sizeof( MEMORY_POOL_KERNEL_BLOCK_PART_VERIFICATION_STORE_DIRECTORY ) + 128 ] = 0; 

    memcpy(
        _file_path,
        MEMORY_POOL_KERNEL_BLOCK_PART_VERIFICATION_STORE_DIRECTORY,
        sizeof( MEMORY_POOL_KERNEL_BLOCK_PART_VERIFICATION_STORE_DIRECTORY ) - 1
    );

    void* _hash_hex = 
        utils::convert_bytes_hex( __block_part_data, 64 );

    memcpy(
        _file_path + sizeof( MEMORY_POOL_KERNEL_BLOCK_PART_VERIFICATION_STORE_DIRECTORY ) - 1,
        _hash_hex,
        128
    ); free( _hash_hex );

    // Checks if there is already the file with given block part hash data
    if ( ! access( _file_path, F_OK ) ) { std::cout << "Block part already in file" << std::endl; return; }

    printf("Block part received in CPU cuda\n");

    for (
        int _ = 0;
        _ < MEMORY_POOL_KERNEL_BLOCK_PART_VERIFICATION_BLOCKS_TIMES_BLOCK_THREADS;
        _++
    ) {

        // Thread was available
        if ( block_part_verification_thread_ready[ _ ].try_acquire() ) {

            hipMallocManaged( block_part_verification_data + _, __block_part_data_size + 8 ); utils::cuda::check_cuda_error();
    
            memcpy(
                block_part_verification_data[ _ ],
                &__block_part_data_size,
                8
            );

            memcpy(
                block_part_verification_data[ _ ] + 8,
                __block_part_data,
                __block_part_data_size
            );

            block_part_verification_semaphores[ _ ].release();

        }
    
    }

}

void memory_pool::cuda::handle_transaction( void* __transaction_data ) {

    for (
        int _ = 0;
        _ < MEMORY_POOL_KERNEL_TRANSACTION_VERIFICATION_BLOCKS_TIMES_BLOCK_THREADS;
        _++
    ) {

        // Thread was available
        if ( transaction_verification_thread_ready[ _ ].try_acquire() ) {

            // Copies the transaction data into device memory
            memcpy(
                transaction_verification_data + _ * (TRANSACTION_PROPAGATION_LENGTH),
                __transaction_data,
                TRANSACTION_PROPAGATION_LENGTH
            );

            transaction_verification_semaphores[ _ ].release(); break;

        }

    }

}

void memory_pool::cuda::transactions_to_broudcast( p2p::Connection** __connections, uint16_t __connections_count ) {

    p2p::Propagation_Protocol* _propagation_protocol = 
        ( p2p::Propagation_Protocol* ) malloc( sizeof( p2p::Propagation_Protocol ) );

    p2p::Packet* _packet;

    for (
        int _ = 0;
        _ < *memory_pool_transaction_capacity;
        _++
    ) {
        
        // Thread was available
        if ( memory_pool_data_broudcast_sems[ _ ].try_acquire() ) {
            
            new ( _propagation_protocol ) 
                p2p::Propagation_Protocol(
                    P2P_PROTOCOLS_PROPAGATION_PROTOCOL_DEFINITIONS_PROPAGATION_TYPE_TRANSACTION,
                    TRANSACTION_PROPAGATION_LENGTH,
                    memory_pool + _ * (TRANSACTION_PROPAGATION_LENGTH)
                );

            _packet = 
                _propagation_protocol->get_packet();

            for ( uint16_t _ = 0; _ < __connections_count; _++ )

                if ( __connections[ _ ]  ) __connections[ _ ]->send_packet( _packet );

        }

    }

}

void* memory_pool::cuda::get_transaction_data_from_memory_pool( uint32_t* __transactions_count ) {

    uint64_t _ready_transactions = *ready_transactions_count;

    *__transactions_count = _ready_transactions;

    if ( ! *__transactions_count ) return 0;

    void* _data = 
        malloc( _ready_transactions * (TRANSACTION_LENGTH) ), *_rtr = _data;

    std::cout << "Malloc pointer: " << _data << std::endl;
    std::cout << "Ready transactions: " << *ready_transactions_count << std::endl;

    for ( uint64_t _ = 0; _ready_transactions && _ < *memory_pool_transaction_capacity; _++ )

        // Have data
        if ( ! memory_pool_sems[ _ ].try_acquire() ) {

            memset(
                _data,
                0,
                TRANSACTION_LENGTH
            );

            memcpy(
                _data,
                memory_pool + _ * (TRANSACTION_PROPAGATION_LENGTH),
                TRANSACTION_PROPAGATION_LENGTH
            );
            
            _data = _data + TRANSACTION_LENGTH;

            memory_pool_sems[ _ ].release();

            (*ready_transactions_count)--; 
            
            _ready_transactions--;

        }

        // No data
        else memory_pool_sems[ _ ].release();

    return _rtr;

}

void memory_pool::cuda::store_broudcast_block_part_data( p2p::Connection** __connections, uint16_t __connections_count ) {

    p2p::Propagation_Protocol* _propagation_protocol = 
        ( p2p::Propagation_Protocol* ) malloc( sizeof( p2p::Propagation_Protocol ) );

    p2p::Packet* _packet = 0;

    char* _file_path = 
        ( char* ) malloc( sizeof( MEMORY_POOL_KERNEL_BLOCK_PART_VERIFICATION_STORE_DIRECTORY ) + 129 );

    void* _hash_hex;

    memcpy(
        _file_path,
        MEMORY_POOL_KERNEL_BLOCK_PART_VERIFICATION_STORE_DIRECTORY,
        sizeof( MEMORY_POOL_KERNEL_BLOCK_PART_VERIFICATION_STORE_DIRECTORY ) - 1
    );

    _file_path[ sizeof( MEMORY_POOL_KERNEL_BLOCK_PART_VERIFICATION_STORE_DIRECTORY ) + 128 ] = 0; 

    for (
        uint32_t _ = 0;
        _ < MEMORY_POOL_KERNEL_BLOCK_PART_VERIFICATION_BLOCK_PART_STORE_FOR_FILE_MEMORY;
        _++
    ) {

        if ( ! block_part_memory_pool_semaphores[ _ ].try_acquire() ) {

            new ( _propagation_protocol ) 
                p2p::Propagation_Protocol(
                    P2P_PROTOCOLS_PROPAGATION_PROTOCOL_DEFINITIONS_PROPAGATION_TYPE_BLOCK_PART,
                    *( uint64_t* ) block_part_memory_pool[ _ ],
                    block_part_memory_pool[ _ ] + 8
                );
            
            _packet = _propagation_protocol->get_packet();

            for ( uint16_t _ = 0; _ < __connections_count; _++ ) if ( __connections[ _ ]  ) __connections[ _ ]->send_packet( _packet );

            _hash_hex = 
                utils::convert_bytes_hex( block_part_memory_pool[ _ ] + 8, 64 );

            memcpy(
                _file_path + sizeof( MEMORY_POOL_KERNEL_BLOCK_PART_VERIFICATION_STORE_DIRECTORY ) - 1,
                _hash_hex,
                128
            ); free( _hash_hex );
  
            utils::write_file_data(
                block_part_memory_pool[ _ ] + 8,
                *( uint64_t* ) block_part_memory_pool[ _ ],
                _file_path
            );

            block_part_memory_pool_semaphores[ _ ].release();

        }

        else block_part_memory_pool_semaphores[ _ ].release();

    }

    free( _file_path );

    _propagation_protocol->~Propagation_Protocol(); free( _propagation_protocol );

    free( _packet );

}


