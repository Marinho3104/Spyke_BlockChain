

#include "kernel_transaction_verification.cuh" // Transaction verification variables
#include "kernel_communication.cuh" // Kernel communication functions
#include "transaction_definitions.h" // Transaction definitions
#include "memory_pool_initialization.cuh"
#include "connection.h" // Struct Connection
#include "propagation_protocol_definitions.h"
#include "propagation_protocol.h" // Struct Propagation Protocol
#include "packet.h" // Struct Packet

// Nvcc libs
#include <cuda/semaphore>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

void memory_pool::cuda::handle_transaction( void* __transaction_data ) {

    for (
        int _ = 0;
        _ < MEMORY_POOL_KERNEL_TRANSACTION_VERIFICATION_BLOCKS_TIMES_BLOCK_THREADS;
        _++
    ) {

        // Thread was available
        if ( transaction_verification_thread_ready[ _ ].try_acquire() ) {

            // Copies the transaction data into device memory
            memcpy(
                transaction_verification_data + _ * TRANSACTION_PROPAGATION_LENGTH,
                __transaction_data,
                TRANSACTION_PROPAGATION_LENGTH
            );

            transaction_verification_semaphores[ _ ].release(); break;

        }

    }

}

void memory_pool::cuda::transactions_to_broudcast( p2p::Connection** __connections, uint16_t __connections_count ) {

    p2p::Propagation_Protocol* _propagation_protocol = 
        ( p2p::Propagation_Protocol* ) malloc( sizeof( p2p::Propagation_Protocol ) );

    p2p::Packet* _packet;

    for (
        int _ = 0;
        _ < *memory_pool_transaction_capacity;
        _++
    ) {
        
        // Thread was available
        if ( memory_pool_data_broudcast_sems[ _ ].try_acquire() ) {
            
            new ( _propagation_protocol ) 
                p2p::Propagation_Protocol(
                    P2P_PROTOCOLS_PROPAGATION_PROTOCOL_DEFINITIONS_PROPAGATION_TYPE_TRANSACTION,
                    TRANSACTION_PROPAGATION_LENGTH,
                    memory_pool + _ * TRANSACTION_PROPAGATION_LENGTH
                );

            _packet = 
                _propagation_protocol->get_packet();

            for ( uint16_t _ = 0; _ < __connections_count; _++ )

                if ( __connections[ _ ]  ) __connections[ _ ]->send_packet( _packet );

        }

    }

}

