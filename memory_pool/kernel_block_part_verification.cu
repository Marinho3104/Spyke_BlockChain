#include "hip/hip_runtime.h"


#include "kernel_block_part_verification.cuh"


// Nvcc libs
#include <cuda/semaphore>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

namespace memory_pool::cuda {

    void** block_part_verification_data;

    ::cuda::std::binary_semaphore* block_part_verification_semaphores;
    ::cuda::std::binary_semaphore* block_part_verification_thread_ready;

    hipStream_t block_part_stream;

}

__global__ void memory_pool::cuda::kernel_block_part_verification( void** __data, ::cuda::std::binary_semaphore* __sems, ::cuda::std::binary_semaphore* __ready_thread, void** __memory_pool_data, ::cuda::std::binary_semaphore* __memory_pool_sems, ::cuda::std::binary_semaphore* __memory_pool_broudcast_sems ) {

    int _global_id = 
        blockIdx.x * blockDim.x + threadIdx.x;

    while( 1 ) {

        __sems[ _global_id ].acquire();

        // Makes all verifications

        // Add to memory_pool 
        kernel_block_part_store_block_part( __data[ _global_id ], __memory_pool_data, __memory_pool_sems, __memory_pool_broudcast_sems );

        __ready_thread[ _global_id ].release();

    }

}

__device__ void memory_pool::cuda::kernel_block_part_store_block_part( void* __block_part_data, void** __memory_pool, ::cuda::std::binary_semaphore* __memory_pool_sems, ::cuda::std::binary_semaphore* __memory_pool_broudcast_sems ) {

    for (
        int _ = 0; 
        _ < MEMORY_POOL_KERNEL_BLOCK_PART_VERIFICATION_BLOCK_PART_STORE_FOR_FILE_MEMORY;
        _++
    ) {

        if ( __memory_pool_sems[ _ ].try_acquire() ) {

            memcpy(
                __memory_pool + _,
                &__block_part_data,
                sizeof( void* )
            );


            __memory_pool_broudcast_sems[ _ ].release();

            printf("Block part stored\n"); break;

        }

    }

}
