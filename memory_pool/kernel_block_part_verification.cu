#include "hip/hip_runtime.h"


#include "kernel_block_part_verification.cuh"


// Nvcc libs
#include <cuda/semaphore>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

namespace memory_pool::cuda {

    void** block_part_verification_data;

    ::cuda::std::binary_semaphore* block_part_verification_semaphores;

    hipStream_t block_part_stream;

}

__global__ void memory_pool::cuda::kernel_block_part_verification( void** __data, ::cuda::std::binary_semaphore* __sems ) {

    int _global_id = 
        blockIdx.x * blockDim.x + threadIdx.x;

    printf("Kernel block part verification launched");

    __sems[ _global_id ].acquire();

    printf("Kernel block part acquired");

}
