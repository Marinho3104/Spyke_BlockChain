

#include "kernel_block_part_verification.cuh"
#include "memory_pool_initialization.cuh"
#include "utils_functions_cuda.cuh"
#include "blockchain_utils.h"

// Nvcc libs
#include <cuda/semaphore>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

namespace memory_pool::cuda {

    ::cuda::std::binary_semaphore* memory_pool_sems;
    ::cuda::std::binary_semaphore* memory_pool_data_broudcast_sems;
    void* memory_pool;

    uint64_t* ready_transactions_count;
    uint64_t* memory_pool_transaction_capacity;

    unsigned char *public_key_type;

    bool *public_key_type_enable;

    unsigned char* block_division;

    uint32_t* size_span_block_division;


    void** block_part_memory_pool;
    ::cuda::std::binary_semaphore* block_part_memory_pool_broudcast_semaphores;
    ::cuda::std::binary_semaphore* block_part_memory_pool_semaphores;

}

void memory_pool::cuda::initialize_memory_pool( uint64_t __memory_pool_size, void* __public_key, unsigned char __block_division, uint32_t __size_span_block_division ) {

    hipMallocManaged( &memory_pool_transaction_capacity, sizeof( uint64_t ) ); utils::cuda::check_cuda_error();
    hipMallocManaged( &ready_transactions_count, sizeof( uint64_t ) ); utils::cuda::check_cuda_error();

    hipMallocManaged( &block_division, sizeof( unsigned char ) ); utils::cuda::check_cuda_error();
    hipMallocManaged( &size_span_block_division, sizeof( uint64_t ) ); utils::cuda::check_cuda_error();

    hipMallocManaged( &public_key_type, sizeof( unsigned char ) ); utils::cuda::check_cuda_error();
    hipMallocManaged( &public_key_type_enable, sizeof( bool ) ); utils::cuda::check_cuda_error();

    *memory_pool_transaction_capacity = __memory_pool_size; *public_key_type_enable = __public_key;

    *block_division = __block_division; *size_span_block_division = __size_span_block_division;

    if ( *public_key_type_enable )

        *public_key_type = 
            blockchain::get_public_key_type( __public_key );

    // Allocate memory for transaction data be stored 
    hipMallocManaged( &memory_pool, __memory_pool_size * TRANSACTION_PROPAGATION_LENGTH ); utils::cuda::check_cuda_error();

    // Allocated memory for memory pool semaphores
    hipMallocManaged( &memory_pool_sems, sizeof( ::cuda::std::binary_semaphore ) * __memory_pool_size ); utils::cuda::check_cuda_error();
    hipMallocManaged( &memory_pool_data_broudcast_sems, sizeof( ::cuda::std::binary_semaphore ) * __memory_pool_size ); utils::cuda::check_cuda_error();

    for ( uint64_t _ = 0; _ < __memory_pool_size; _++ ) { new ( memory_pool_sems + _ ) ::cuda::std::binary_semaphore( 1 ); new ( memory_pool_data_broudcast_sems + _ ) ::cuda::std::binary_semaphore( 0 ); }

    hipMallocManaged( &block_part_memory_pool, MEMORY_POOL_KERNEL_BLOCK_PART_VERIFICATION_BLOCK_PART_STORE_FOR_FILE_MEMORY * sizeof( void* ) ); utils::cuda::check_cuda_error();

    for ( uint64_t _ = 0; _ < MEMORY_POOL_KERNEL_BLOCK_PART_VERIFICATION_BLOCK_PART_STORE_FOR_FILE_MEMORY; _++ ) block_part_memory_pool[ _ ] = 0;

    hipMallocManaged( &block_part_memory_pool_semaphores, sizeof( ::cuda::std::binary_semaphore ) * MEMORY_POOL_KERNEL_BLOCK_PART_VERIFICATION_BLOCK_PART_STORE_FOR_FILE_MEMORY ); 
    hipMallocManaged( &block_part_memory_pool_broudcast_semaphores, sizeof( ::cuda::std::binary_semaphore ) * MEMORY_POOL_KERNEL_BLOCK_PART_VERIFICATION_BLOCK_PART_STORE_FOR_FILE_MEMORY ); 
    utils::cuda::check_cuda_error();

    for ( uint64_t _ = 0; _ < MEMORY_POOL_KERNEL_BLOCK_PART_VERIFICATION_BLOCK_PART_STORE_FOR_FILE_MEMORY; _++ )

        { new ( block_part_memory_pool_semaphores + _ ) ::cuda::std::binary_semaphore( 1 ); new ( block_part_memory_pool_broudcast_semaphores + _ ) ::cuda::std::binary_semaphore( 0 ); }

        

}

