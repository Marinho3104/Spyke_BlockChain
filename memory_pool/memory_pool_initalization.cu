

#include "memory_pool_initialization.cuh"
#include "utils_functions_cuda.cuh"

// Nvcc libs
#include <cuda/semaphore>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

namespace memory_pool::cuda {

    ::cuda::std::binary_semaphore* memory_pool_sems;
    ::cuda::std::binary_semaphore* memory_pool_data_broudcast_sems;
    void* memory_pool;

    uint64_t* ready_transactions_count;
    uint64_t* memory_pool_transaction_capacity;

    unsigned char *public_key_type;

    bool *public_key_type_enable;

}

void memory_pool::cuda::initialize_memory_pool( uint64_t __memory_pool_size ) {

    hipMallocManaged( &memory_pool_transaction_capacity, sizeof( uint64_t ) ); utils::cuda::check_cuda_error();
    hipMallocManaged( &ready_transactions_count, sizeof( uint64_t ) ); utils::cuda::check_cuda_error();

    hipMallocManaged( &public_key_type, sizeof( unsigned char ) ); utils::cuda::check_cuda_error();
    hipMallocManaged( &public_key_type_enable, sizeof( bool ) ); utils::cuda::check_cuda_error();

    *memory_pool_transaction_capacity = __memory_pool_size; *public_key_type_enable = 0;

    // Allocate memory for transaction data be stored 
    hipMallocManaged( &memory_pool, __memory_pool_size * TRANSACTION_PROPAGATION_LENGTH ); utils::cuda::check_cuda_error();

    // Allocated memory for memory pool semaphores
    hipMallocManaged( &memory_pool_sems, sizeof( ::cuda::std::binary_semaphore ) * __memory_pool_size ); utils::cuda::check_cuda_error();
    hipMallocManaged( &memory_pool_data_broudcast_sems, sizeof( ::cuda::std::binary_semaphore ) * __memory_pool_size ); utils::cuda::check_cuda_error();

    for ( uint64_t _ = 0; _ < __memory_pool_size; _++ ) { new ( memory_pool_sems + _ ) ::cuda::std::binary_semaphore( 1 ); new ( memory_pool_data_broudcast_sems + _ ) ::cuda::std::binary_semaphore( 0 ); }

}

