

#include "memory_pool_initialization.cuh"
#include "utils_functions_cuda.cuh"

// Nvcc libs
#include <cuda/semaphore>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

namespace memory_pool::cuda {

    ::cuda::std::binary_semaphore* memory_pool_sems;
    void* memory_pool;

    uint64_t* ready_transactions_count;
    uint64_t* memory_pool_transaction_capacity;

}

void memory_pool::cuda::initialize_memory_pool( uint64_t __memory_pool_size ) {

    hipMalloc( &memory_pool_transaction_capacity, sizeof( uint64_t ) ); utils::cuda::check_cuda_error();
    hipMalloc( &ready_transactions_count, sizeof( uint64_t ) ); utils::cuda::check_cuda_error();

    hipMemcpy(
        memory_pool_transaction_capacity,
        &__memory_pool_size,
        sizeof( uint64_t ),
        hipMemcpyHostToDevice
    ); utils::cuda::check_cuda_error();

    // Allocate memory for transaction data be stored 
    hipMalloc( &memory_pool, __memory_pool_size * TRANSACTION_PROPAGATION_LENGTH ); utils::cuda::check_cuda_error();

    // Allocated memory for memory pool semaphores
    hipMalloc( &memory_pool_sems, sizeof( ::cuda::std::binary_semaphore ) * __memory_pool_size ); utils::cuda::check_cuda_error();

    ::cuda::std::binary_semaphore* _bin_sems = 
        ( ::cuda::std::binary_semaphore* ) malloc( sizeof( ::cuda::std::binary_semaphore ) * __memory_pool_size );

    for ( uint64_t _ = 0; _ < __memory_pool_size; _++ ) new ( _bin_sems + _ ) ::cuda::std::binary_semaphore( 1 );

    hipMemcpy(
        memory_pool_sems,
        _bin_sems,
        sizeof( ::cuda::std::binary_semaphore ) * __memory_pool_size,
        hipMemcpyHostToDevice
    ); utils::cuda::check_cuda_error();

    free( _bin_sems );

}

