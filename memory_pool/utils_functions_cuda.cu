#include <hip/hip_runtime.h>
#include <stdio.h>

#include "utils_functions_cuda.cuh"

void utils::cuda::check_cuda_error() {

    hipError_t _error = hipGetLastError();

    if (
        _error != hipSuccess
    ) { printf("Cuda Error: %s", hipGetErrorString( _error ) ); exit( 1 ); }

}

